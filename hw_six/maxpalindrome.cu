// Elapsed Real Time for input-4.txt: 0m1.376s

/** 
    @file maxpalindrome.cu
    @author Ian M Brain (imbrain)
    This program determines the longest palindrome from an input file containing a sequence of characters.
    Utilizes CUDA programming to run palindrome operations on the gpu.
    Users can specify report flag to indicate the maximum palindrome found by each thread.
*/

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of letters.
char *vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of letters.
int vCap = 0;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: maxpalindrome [report]\n" );
  exit( 1 );
}

// Read the list of letters.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (char *) malloc( vCap * sizeof( char ) );

  // Keep reading as many values as we can.
  char v;
  while ( scanf( "%c\n", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (char *) realloc( vList, vCap * sizeof( char ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

// Determines the length of the longest palindrome for the characters the thread is responsible for.
// Runs on the gpu uitilizing CUDA programming.
__global__ void checkLength( int vCount, bool report, char *dList, int *lengthList ) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if ( idx < vCount ) {
    // ...

    // Index tracking the current character from the end of the list.
    int countDown = 0;

    // Indicates if there is no palindrome and thus the max palindrome should not be set.
    bool noPalindrome = false;

    // Length of the longest palindrome.
    int maxPalindrome = 1;

    /** Citing Help from MaxPalindrome.c that I completed on 2/1/2024
    * The code for this method is based on the main method operation to parse through input file characters to find palindromes from MaxPalindrome.c
    */
    for ( int i = idx + 1; i < vCount; i++ ) {
      noPalindrome = false;

      countDown = 0;

      // Start at the thread index and parse every character sequence starting from the index to the character at index vCount.
      for ( int j = idx; j < ( i - countDown ); j++ ) {

        // If the characters are not equal, set noPalindrom to true and stop comparing the characters.
        if( dList[ j ] != dList[ i - countDown ] ) {
          noPalindrome = true;
          break;
        } 

        countDown++;
      }

      // If there is a palindrome and the current palindrome is longer than the last longest palindrome, set the longest palindrome to this length.
      if ( !noPalindrome && maxPalindrome < ( i - idx + 1 ) ) {
        maxPalindrome = ( i - idx + 1 );
      }
    }

    lengthList[ idx ] = maxPalindrome;
  }
}

int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();

  // Add code to allocate memory on the device and copy over the list.

  // List on the device that will hold the copied values from the host input files.
  char *dList;

  /* Citing Help from Course Website
  * The code for allocating device memory is based on slide "Preparing Device Input" from the "Intro to CUDA Programming" lecture.
  */
  hipMalloc( ( void ** ) &dList, vCount * sizeof( char ) );

  // Add code to copy the list over to the device.

  /* Citing Help from Course Website
  * The code for copying host memory to the device is based on slide "Preparing Device Input" from the "Intro to CUDA Programming" lecture.
  */
  // Copy the list of characters from the host to the memory in device.
  hipMemcpy( dList, vList, vCount * sizeof( char ), hipMemcpyHostToDevice );

  // Add code to allocate space on the device to hold the results.

  // List on the device that will hold lengths of palindromes
  int *lengthList;

  /* Citing Help from Course Website
  * The code for allocating device memory is based on slide "Preparing Device Input" from the "Intro to CUDA Programming" lecture.
  */
  hipMalloc( ( void ** ) &lengthList, vCount * sizeof( int ) );
  

  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions (you'll need to add some parameters)
  checkLength<<<blocksPerGrid, threadsPerBlock>>>( vCount, report, dList, lengthList );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Add code to copy results back to the host, compare the local maximum lengths
  // and report the final maximum length

  // Allocate memory for the palindrome lengths on the host.
  int *results = ( int * ) malloc( vCount * sizeof( int ) );

  /* Citing Help from Course Website
  * The code for copying device memory to the host is based on slide "Recovering Device Output" from the "Intro to CUDA Programming" lecture.
  */
  // Copy the palindrome length memory from the device to the host.
  hipMemcpy( results, lengthList, vCount * sizeof( int ), hipMemcpyDeviceToHost );

  // Maximum length of all the palindromes.
  int maxLength = 1;

  // For each thread, determine the maximum length of all the threads and report the max length of that thread if the report argument is specified.
  for ( int i = 0; i < vCount; i++ ) {
    if ( results[ i ] > maxLength ) {
      maxLength = results[ i ];
    }

    if ( report ) {
      printf( "I’m thread %d. Max length found: %d.\n", i, results[ i ] );
    }
  }

  // Print the maximum length of all the threads
  printf( "Maximum Length: %d\n", maxLength );

  

  // Free memory on the device and the host.
  // ...
  free( vList );
  free( results );
  hipFree( dList );
  hipFree( lengthList );

  hipDeviceReset();

  return 0;
}
